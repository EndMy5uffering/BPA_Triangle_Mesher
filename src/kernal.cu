// kernel.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernel() {
    printf("Hello from the GPU thread %d\n", threadIdx.x);
}

extern "C" void launchKernel() {
    myKernel<<<1, 10>>>();
    hipDeviceSynchronize(); // Wait for the GPU to finish
}
